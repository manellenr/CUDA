#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 1000000
#define BLOCK_SIZE 256

__device__ void permuter(int *a, int *b) {
    int tmp = *a;
    *a = *b;
    *b = tmp;
}

__device__ int partition(int *arr, int low, int high) {
    int pivot = arr[high];
    int i = (low - 1);
    for (int j = low; j < high; j++) {
        if (arr[j] <= pivot) {
            i++;
            permuter(&arr[i], &arr[j]);
        }
    }
    permuter(&arr[i + 1], &arr[high]);
    return (i + 1);
}

__global__ void quickSortKernel(int *arr, int low, int high) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (low < high) {
        int pivotIndex = partition(arr, low, high);

        if (pivotIndex - 1 > low && pivotIndex - 1 > idx) {
            quickSortKernel<<<1, BLOCK_SIZE>>>(arr, low, pivotIndex - 1);
        }
        if (pivotIndex + 1 < high && pivotIndex + 1 < idx) {
            quickSortKernel<<<1, BLOCK_SIZE>>>(arr, pivotIndex + 1, high);
        }
    }
}

int main() {
    int *arr, *d_arr;
    int n = N;
    srand(time(NULL));

    arr = (int *)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) {
        arr[i] = rand() % 1000000 + 1;
    }

    hipMalloc((void**)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    quickSortKernel<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_arr, 0, n - 1);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU execution time = %f s\n", milliseconds / 1000);

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array:\n");
    for (int i = 0; i < n; i++) {
        printf(" %d", arr[i]);
    }
    printf("\n");

    free(arr);
    hipFree(d_arr);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
